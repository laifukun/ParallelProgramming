#include "hip/hip_runtime.h"

#include "kmeans.h"
#include <stdio.h>
#define TPB 32


__device__ __host__ DataType cudaDistance(DataType* v1, DataType* v2, int d) {
    DataType dist = 0.0;
    for (int i = 0; i < d; i++) {
        dist += (*(v1+i) - *(v2+i))*(*(v1+i) - *(v2+i));
    }
    return sqrt(dist);
}


__host__ bool isConverged(DataType* curCentriod, DataType* prevCentriod, DataType threshold, int n_centroids, int dim) {
    //int idx = blockIdx.x*blockDim.x + threadIdx.x;
    for (int i = 0; i < n_centroids; i++)
        if (cudaDistance(curCentriod + i*dim, prevCentriod + i*dim, dim) >= threshold) return false;
    return true;
}



__global__ void assign_centroid(DataType* points, DataType* centroids, int* clusterIds, int n_points, int n_centroids, int dims)
 {
    const int idx = blockIdx.x*blockDim.x + threadIdx.x;
    //printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
    if (idx >= n_points) return;

    DataType minDist = INFINITY;

    for (int j = 0; j < n_centroids; j++) {
        DataType dist = cudaDistance(centroids + j*dims, points + idx*dims, dims);
        //printf("dist: %lf, minDist: %lf, clusterIds: %d, j: %d, points: %lf, centroids: %lf\n", dist, minDist, idx, j, *(points+idx*dims), *(centroids + j*dims)); 
        if ( dist < minDist) {
            minDist = dist;
            clusterIds[idx] = j;              
        }            
    }
    
    //clusterIds[idx] = idx;
}

__global__ void assign_centroid_shared(DataType* points, DataType* centroids, int* clusterIds, int n_points, int n_centroids, int dims)
 {
    //const int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (blockIdx.x >= n_points) return;

    const int threadId = threadIdx.x;
    //extern __shared__ char localMemory[];
    __shared__ double minDist[1];
    
    //DataType* minDist = (DataType*)&localMemory;
    //int* minIdx = (int*)&localMemory[sizeof(DataType)];
    minDist[0] = INFINITY;
    
    __syncthreads();

    
    //for (int j = 0; j < n_centroids; j++) {
    DataType dist = cudaDistance(centroids + threadId*dims, points + blockIdx.x*dims, dims);

    atomicMin((unsigned long long int*)minDist, __double_as_longlong(dist));
        
    __syncthreads();
    
    if (__double_as_longlong(dist) == __double_as_longlong(*minDist)) {
        clusterIds[blockIdx.x] = threadId;
    }

    printf("Block: %d, thread: %d, dist: %lf, minDist: %lf, minCluster: %d\n", blockIdx.x, threadId, dist, *minDist, clusterIds[blockIdx.x]);
    //clusterIds[idx] = idx;
}

__global__ void group_centroid(DataType* points, DataType* centroids,  int* cluster_of_points, int* cluster_size, int n_points, int n_centroids, int dims) {


    const int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx >= n_points) return;
        

    const int clusterId = cluster_of_points[idx];
    for (int j = 0; j < dims; j++) {
        atomicAdd(centroids+clusterId*dims + j, *(points + idx*dims + j));  
        //printf("idx: %d, j: %d, points: %lf, centroids: %lf\n", idx, j, points[idx*dims + j], centroids[clusterId*dims + j]); 
    }
    atomicAdd(&cluster_size[clusterId], 1);
    //printf("idx: %d, clusterId: %d, cluster size: %d\n", idx, clusterId, cluster_size[clusterId]);
    //std::cout<<points_centroid[clusterIds[i]]<<std::endl;

    //__syncthreads();

/*
    if (idx < n_centroids && cluster_size[idx] != 0) {
        centroids[idx] = centroids[idx] / cluster_size[idx] ;
    }
    */

}

__global__ void update_centoid(DataType* centroids, int* cluster_size, int n_centroids, int dims) {


    const int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx < n_centroids && cluster_size[idx] != 0) {
        for (int j = 0; j < dims; j++) {
            centroids[idx*dims + j] /= cluster_size[idx] ;
        }        
    }
    
}


__global__ void group_centroid_shared(DataType* points, DataType* centroids,  int* cluster_of_points, int* cluster_size, int n_points, int n_centroids, int dims) {


    const int idx = blockIdx.x*blockDim.x + threadIdx.x;
    
    if (idx >= n_points) return;
    

    const int threadId = threadIdx.x;
    extern __shared__ int localMemory[];
    
    DataType* block_points = (DataType*)&localMemory;
    printf("Hello from blockId %d, threadId %d, blockDim %d\n", blockIdx.x, threadIdx.x, blockDim.x);
    for (int d = 0; d < dims; d++) {
        block_points[threadId*dims + d] = points[idx*dims + d];
    }
    
    __shared__ int block_cluster_of_points[TPB];
    block_cluster_of_points[threadId] = cluster_of_points[idx];

    
    __syncthreads();

    //int clusterId = block_cluster_of_points[threadId];

    if (threadId == 0) {

        DataType* block_cluster_point_sum = (DataType*)&localMemory[TPB*dims];
        int* block_cluster_size = (int*)&localMemory[TPB*dims + n_centroids*dims];

        for (int i = 0; i < n_centroids; i++) {
            for (int d = 0; d < dims; d++) {                
                block_cluster_point_sum[i*dims + d] = 0.0;                
            }
            block_cluster_size[i] = 0;
        }

        for (int j = 0; j < blockDim.x; j++){
            int clusterId = block_cluster_of_points[j];
            for (int d = 0; d < dims; d++) {                
                block_cluster_point_sum[clusterId*dims + d] += block_points[j*dims + d];                
            }
            block_cluster_size[clusterId] += 1;
        }
            
        
        for (int i = 0; i < n_centroids; i++) {
            for (int d = 0; d < dims; d++)
                atomicAdd(centroids+i*dims + d, block_cluster_point_sum[i*dims+d]);
            atomicAdd(&cluster_size[i], block_cluster_size[i]);
        }
    }

    
    //std::cout<<points_centroid[clusterIds[i]]<<std::endl;

    //__syncthreads();

    //printf("idx: %d, clusterId: %d, cluster size: %d\n", idx, clusterId, cluster_size[clusterId]);
    /*
    if (idx < n_centroids && cluster_size[idx] != 0) {
        for (int j = 0; j < dims; j++) {
            centroids[idx*dims + j] /= cluster_size[idx] ;
        }        
    }
    */

}

void kmean_cuda(kmean_args_t* args, double* timeMeasure, int algo){

    hipEvent_t start, stop, assignStart, assignEnd, updateStart, updateEnd;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&assignStart);
    hipEventCreate(&assignEnd);
    hipEventCreate(&updateStart);
    hipEventCreate(&updateEnd);
    hipEventRecord(start);

    int n_vals = args->n_vals;
    int n_centroids = args->num_cluster;
    int dims = args->dims;
    args->iter_converge = 0;

    DataType* d_points=0, *d_centroids=0;
    int* cluster_id=0, *cluster_size=0;

    hipMalloc(&d_points, n_vals*dims*sizeof(DataType));
    hipMalloc(&d_centroids, n_centroids*dims*sizeof(DataType));
    hipMalloc((void**)&cluster_id, n_vals*sizeof(int));
    hipMalloc(&cluster_size, n_centroids*sizeof(int));

    hipMemcpy(d_points, args->input_vals, n_vals*dims*sizeof(DataType), hipMemcpyHostToDevice);
    hipMemcpy(d_centroids, args->centroids, n_centroids*dims*sizeof(DataType), hipMemcpyHostToDevice);

    DataType* tmpCentroid = (DataType*) malloc(n_centroids* dims * sizeof(DataType));
    hipEventRecord(start);
    //std::cout<<"Iter: " <<args->iter_converge<<std::endl;
    while (args->iter_converge < args->max_num_iter && !isConverged(args->centroids, tmpCentroid, args->threshold, n_centroids, dims)) {
        args->iter_converge++;     
        //copyCentroids(tmpCentroid, args->centroids, args->num_cluster, args->dims);
        memcpy(tmpCentroid, args->centroids, n_centroids*dims*sizeof(DataType));
        hipEventRecord(assignStart);

        if (algo == 0) {
            //assign_centroid<<<(n_vals+TPB-1)/TPB,TPB>>>(d_points, d_centroids,  cluster_id, n_vals, n_centroids, dims);
            assign_centroid<<<(n_vals*n_centroids+n_centroids-1)/n_centroids,n_centroids>>>(d_points, d_centroids,  cluster_id, n_vals, n_centroids, dims);
        } else if (algo == 1) {
            assign_centroid_shared<<<(n_vals*n_centroids+n_centroids-1)/n_centroids,n_centroids>>>(d_points, d_centroids,  cluster_id, n_vals, n_centroids, dims);
        }

        hipEventRecord(assignEnd);
        hipEventSynchronize(assignEnd);
        float tmp1 = 0;
        hipEventElapsedTime(&tmp1, assignStart, assignEnd);
        timeMeasure[0] += tmp1;
        //add<<<N,1>>>(cluster_id);
        
        //hipMemcpy(args->centroids, d_centroids, n_centroids*dims*sizeof(DataType), hipMemcpyDeviceToHost);
        //hipMemcpy(args->clusterId_of_point, cluster_id, n_vals*sizeof(int), hipMemcpyDeviceToHost);

        //std::cout<<args->iter_converge<<std::endl;m
        //std::cout<<std::endl<<"Iter: " <<args->iter_converge<<std::endl;
        //hipDeviceSynchronize();
        /*
        for (int i = 0; i < n_vals; i++){
            printf("%d ", args->clusterId_of_point[i]);
        }
        printf("\n");
        */
        /*
        for (int clusterId = 0; clusterId < args->num_cluster; clusterId++){
            printf("%d ", clusterId);
            for (int d = 0; d < args->dims; d++)
                printf("%lf ", args->centroids[clusterId*args->dims + d]);
        printf("\n");
        }
        */
        hipMemset(d_centroids, 0.0, n_centroids*dims*sizeof(DataType));
        hipMemset(cluster_size, 0, n_centroids*sizeof(int));

        hipEventRecord(updateStart);

        if (algo == 0) {
            group_centroid<<<(n_vals+TPB-1)/TPB,TPB>>>(d_points, d_centroids,  cluster_id, cluster_size, n_vals, n_centroids, dims);
        } else if (algo == 1) {
            group_centroid_shared<<<(n_vals+TPB-1)/TPB,TPB, (TPB+n_centroids)*dims*sizeof(DataType) + n_centroids*sizeof(int)>>>(d_points, d_centroids,  cluster_id, cluster_size, n_vals, n_centroids, dims);

        }        
        
        update_centoid<<<(n_vals+TPB-1)/TPB,TPB>>>(d_centroids,  cluster_size, n_centroids, dims);
        hipEventRecord(updateEnd);
        hipEventSynchronize(updateEnd);
        float tmp2 = 0;
        hipEventElapsedTime(&tmp2, updateStart, updateEnd);
        timeMeasure[1] += tmp2;

        hipMemcpy(args->centroids, d_centroids, n_centroids*dims*sizeof(DataType), hipMemcpyDeviceToHost);
        
        //bool testCondition = isConverged(args->centroids, tmpCentroid, args->threshold, n_centroids, dims);
        //std::cout<<"Test Condition: "<<testCondition<<std::endl;

        
        for (int clusterId = 0; clusterId < args->num_cluster; clusterId++){
            printf("%d ", clusterId);
            for (int d = 0; d < args->dims; d++)
                printf("%lf ", args->centroids[clusterId*args->dims + d]);
        printf("\n");
        }
        
    }
    hipMemcpy(args->clusterId_of_point, cluster_id, n_vals*sizeof(int), hipMemcpyDeviceToHost);

    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    free(tmpCentroid);
    hipFree(d_points);
    hipFree(d_centroids);
    hipFree(cluster_id);
    hipFree(cluster_size);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float tmp3 = 0;
    hipEventElapsedTime(&tmp3, start, stop);
    timeMeasure[2] += tmp3;
    timeMeasure[3] = timeMeasure[2] - timeMeasure[1] - timeMeasure[0];
}


