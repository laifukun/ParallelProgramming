#include "hip/hip_runtime.h"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include "kmeans.h"
#include "defines.h"


template <typename T>
struct index_to_row : public thrust::unary_function<T,T>
{
  T C; // number of columns
  
  __host__ __device__
  index_to_row(T C) : C(C) {}

  __host__ __device__
  T operator()(T i)
  {
    return i / C;
  }
};

template <typename T>
struct square
{
    __host__ __device__
        T operator()(const T& x) const { 
            return x * x;
        }
};

template <typename T>
struct lessThanThreshold
{
    T threshold; 
  
    __host__ __device__
    lessThanThreshold(T t) : threshold(t) {}

    __host__ __device__
        bool operator()(const T& y) const { 
            printf("y: %e, sqrt(y): %e\n", y, sqrt(y));
            //return y >= threshold*threshold ? false : true;
            return (y >= threshold) ? false : true;
        }
};

template <typename T>
struct squareDiff
{
    __host__ __device__
        T operator()(const T& x, const T& y) const { 
            return (x-y) * (x-y);
        }
};

struct boolAnd
{
    __host__ __device__
        bool operator()(const bool x, const bool y) const { 
            return x&&y;
        }
};

typedef thrust::tuple<DataType, DataType> point_center_tuple;

struct point_distance : public thrust::unary_function<point_center_tuple, DataType>
{
  __host__ __device__ DataType operator()(const point_center_tuple &pc_tuple) const {
    DataType temp = thrust::get<0>(pc_tuple) - thrust::get<1>(pc_tuple);
    return temp*temp;
  }
};


struct point_idx
{
  int dim;
  int n_c;

  point_idx(int _dim, int _n) : dim(_dim), n_c(_n) {};

  __host__ __device__ int operator()(const int idx) const {
      return (idx % dim) + (dim * (idx/(dim*n_c)));    
    }
};

struct center_idx
{
  int dim;
  int n_c;

  center_idx(int _dim, int _n) : dim(_dim), n_c(_n) {};

  __host__ __device__ int operator()(const int idx) const {
    return (idx % (dim*n_c));
    }
};

template <typename T>
struct my_sqrt
{
  __host__ __device__ T operator()(const T val) const {
    return sqrt(val);
  }
};

template<typename T>
struct min_index
{
    __host__ __device__
        T operator()(const T& x, const T& y) const { 
            return (x-y) * (x-y);
        }
};


struct size_accumulator {
  int *r;
  size_accumulator(int *_r) : r(_r) {};
  
  template <typename T>
  __host__ __device__
  void operator()(T t) {
    atomicAdd(r+thrust::get<0>(t), 1);  
  }
};


struct centroid_accumulator {
  DataType *c, *p;
  int dim;
  centroid_accumulator(DataType *_c, DataType *_p, int _dim) : c(_c), p(_p), dim(_dim) {};
  
  template <typename T>
  __host__ __device__
  void operator()(const T& t) {
      //printf("val: %lf, idx: %d \n", t, thrust::get<0>(t), thrust::get<1>(t));
      for (int i = 0; i < dim; i++)
        atomicAdd(&c[thrust::get<0>(t)*dim + i], *(p + thrust::get<1>(t)*dim + i));  
    //atomicAdd(c+thrust::get<0>(t), *(p + thrust::get<1>(t)*dim + i));  
  }
};

template <typename T>
struct centroid_mean {

  __host__ __device__
  T operator()(const T& val, const int& count) {
      //printf("val: %lf count: %d\n", val, count);
      //if (count == 0) return val;
      return val/count;
  }
};

__host__ bool isConverged(thrust::host_vector<DataType>& curCentroid, thrust::host_vector<DataType>& prevCentroid, DataType threshold, int n_centroids, int dim) {
    thrust::host_vector<DataType> point_diff(n_centroids);
    //thrust::host_vector<int> point_idx(n_centroids);
    thrust::host_vector<DataType> tmp(n_centroids*dim);
    //thrust::transform(curCentroid.begin(), curCentroid.end(), prevCentroid.begin(), tmp.begin(), squareDiff<DataType>());
    
    for (int i = 0; i < curCentroid.size(); i++) {
        tmp[i] = (curCentroid[i] - prevCentroid[i]) * (curCentroid[i] - prevCentroid[i]);
    }
    /*
    thrust::reduce_by_key
    (thrust::make_transform_iterator(thrust::counting_iterator<int>(0), index_to_row<int>(dim)),
     thrust::make_transform_iterator(thrust::counting_iterator<int>(n_centroids*dim), index_to_row<int>(dim)),
     tmp.begin(),
     thrust::make_discard_iterator(),
     //point_idx.begin(),
     point_diff.begin(),
     thrust::equal_to<int>(),
     thrust::plus<DataType>());
    */
    thrust::fill(point_diff.begin(), point_diff.end(), 0);
    for (int i = 0; i < point_diff.size(); i++) {
        for (int j = 0; j < dim; j++)
            point_diff[i] +=  tmp[i*dim + j];
        point_diff[i] = sqrt(point_diff[i]);
    }

    //thrust::transform(point_diff.begin(), point_diff.end(), point_diff.begin(), my_sqrt<DataType>());
    lessThanThreshold<DataType> ltThreshold(threshold);
    
    

    for(int i = 0; i < point_diff.size(); i++)
        std::cout << "p[" << i << "] = " << point_diff[i] << std::endl;
    

    return thrust::transform_reduce(point_diff.begin(), point_diff.end(), ltThreshold, true,  boolAnd());
}


void assign_centroid(thrust::device_vector<int>& cluster_point,
                    thrust::device_vector<DataType>& points, 
                    thrust::device_vector<DataType>& centroids, 
                    int n_vals, int dim, int n_centroids) {

        thrust::device_vector<DataType> dist(n_vals*n_centroids);
          thrust::reduce_by_key(
              thrust::make_transform_iterator(thrust::make_counting_iterator<int>(0), index_to_row<int>(dim)), 
              thrust::make_transform_iterator(thrust::make_counting_iterator<int>(dim*n_vals*n_centroids), index_to_row<int>(dim)
                  ),
              thrust::make_transform_iterator(
                thrust::make_zip_iterator(
                    thrust::make_tuple(
                                  thrust::make_permutation_iterator(
                                              points.begin(), 
                                              thrust::make_transform_iterator(
                                                  thrust::make_counting_iterator<int>(0), 
                                                  point_idx(dim, n_centroids)
                                                  )
                                            ),
                                  thrust::make_permutation_iterator(
                                      centroids.begin(), 
                                      thrust::make_transform_iterator(
                                          thrust::make_counting_iterator<int>(0), 
                                          center_idx(dim, n_centroids)
                                          )
                                    )                                    
                                )
                            ), 
                            point_distance()
                        ), 
              thrust::make_discard_iterator(), 
              dist.begin()
            );
    /*
    std::cout<<"size "<< dist.size()<<std::endl;
    std::cout<<std::endl;
    for(int i = 0; i < n_vals; i++){
        for (int j = 0; j < n_centroids; j++)
            std::cout << dist[i*n_centroids+j] <<" ";
        std::cout<<std::endl;
    }
    
    std::cout<<"Centroids"<<std::endl;
    for(int i = 0; i < n_centroids; i++){
        for (int j = 0; j < dim; j++)
            std::cout << centroids[i*dim+j] <<" ";
        std::cout<<std::endl;
    }
    */

/*
    std::cout<<"Points"<<std::endl;
    for(int i = 0; i < n_centroids; i++){
        for (int j = 0; j < dim; j++)
            std::cout << points[i*dim+j] <<" ";
        std::cout<<std::endl;
    }
    */
    thrust::device_vector<DataType> minval(n_vals);
    thrust::reduce_by_key
    (thrust::make_transform_iterator(thrust::counting_iterator<int>(0), index_to_row<int>(n_centroids)),
     thrust::make_transform_iterator(thrust::counting_iterator<int>(n_vals*n_centroids), index_to_row<int>(n_centroids)),
     thrust::make_zip_iterator(
            thrust::make_tuple(
                /*thrust::make_permutation_iterator(
                dist.begin(),
                thrust::make_transform_iterator(
                    thrust::make_counting_iterator<int>(0), 
                    //thrust::placeholders::_1 % n_centroids
                    //(thrust::placeholders::_1 % n_vals) * n_centroids + thrust::placeholders::_1 / n_vals
                    //(thrust::placeholders::_1 % n_centroids) * n_vals + thrust::placeholders::_1 / n_centroids
                    //thrust::placeholders::_1 / n_centroids
                    //point_idx(1, n_centroids)
                    )
                ),*/
                dist.begin(),
                thrust::make_transform_iterator(thrust::make_counting_iterator<int>(0), 
                thrust::placeholders::_1 % n_centroids
                //center_idx(1, n_centroids)                
                )
                )
        ),
     thrust::make_discard_iterator(),
     thrust::make_zip_iterator(
                    thrust::make_tuple(
                            minval.begin(),
                            cluster_point.begin())),
     thrust::equal_to<int>(),
     thrust::minimum<thrust::tuple<DataType, int> >()
     );
    /*
     for(int i = 0; i < cluster_point.size(); i++)
        std::cout << "c[" << i << "] = " << minval[i] << " "<<  cluster_point[i] << std::endl;
        */

}


void update_centroid(thrust::device_vector<DataType>& centroids, 
                    thrust::device_vector<int>& cluster_size,
                    thrust::device_vector<int>& cluster_point,
                    thrust::device_vector<DataType>& points,
                    int n_vals, int dim, int n_centroids) {

    for(int i = 0; i < cluster_point.size(); i++)
        std::cout << "d[" << i << "] = " << cluster_point[i] << std::endl;
    /*
    thrust::transform(thrust::make_permutation_iterator(cluster_size.begin(), cluster_point.begin()), 
        thrust::make_permutation_iterator(cluster_size.begin(), cluster_point.end()), 
        thrust::make_permutation_iterator(cluster_size.begin(), cluster_point.begin()),
        add_one());
    */
   thrust::device_vector<int> indices(n_vals);
   thrust::copy(thrust::counting_iterator<int>(0),
                 thrust::counting_iterator<int>(n_vals),
                 indices.begin());
   thrust::sort_by_key(cluster_point.begin(), cluster_point.end(), indices.begin());
   thrust::reduce_by_key(cluster_point.begin(), cluster_point.end(), thrust::constant_iterator<int>(1), thrust::make_discard_iterator(),
   cluster_size.begin());

   //thrust::sort_by_key(indices.begin(), indices.end(), cluster_size.begin());
    //thrust::for_each_n(thrust::make_zip_iterator(thrust::make_tuple(d_a.begin(), d_b.begin())), size_a, my_func(thrust::raw_pointer_cast(d_r.data())));
    //size_accumulator s_a(thrust::raw_pointer_cast(cluster_size.data()));
    //thrust::for_each_n(thrust::make_zip_iterator(thrust::make_tuple(cluster_point.begin(), thrust::constant_iterator<int>(1))), 
    //            n_vals, s_a);
    /*
    for (int i = 0; i < cluster_point.size(); i++) {
        cluster_size[cluster_point[i]]++; 
    }
    */
   /*
   for(int i = 0; i < cluster_size.size(); i++)
        std::cout << "d[" << i << "] = " << cluster_size[i] << std::endl;


    thrust::reduce_by_key(thrust::make_transform_iterator(thrust::counting_iterator<int>(0), index_to_row<int>(n_centroids)),
                        thrust::make_transform_iterator(thrust::counting_iterator<int>(n_vals*n_centroids), index_to_row<int>(n_centroids)),
        cluster_point.begin(), cluster_point.end(), 
        indices.begin(), 
        thrust::make_discard_iterator(),
        centroids.begin(), thrust::equal<int>(), );    
       */ 
        
    centroid_accumulator c_a(thrust::raw_pointer_cast(centroids.data()), thrust::raw_pointer_cast(points.data()), dim);
    thrust::for_each_n(thrust::make_zip_iterator(thrust::make_tuple(cluster_point.begin(), thrust::make_counting_iterator<int>(0))), 
                n_vals, c_a);
    
    /*
    for (int i = 0; i < points.size()/dim; i++) {
        for (int j = 0; j < dim; j++) {
            //printf("%d, %d\n", i, j);
            centroids[cluster_point[i]*dim + j] += points[i*dim + j];
        }
    }
    */
    thrust::transform(centroids.begin(), centroids.end(), 
        thrust::make_permutation_iterator(
                cluster_size.begin(), 
                thrust::make_transform_iterator(thrust::make_counting_iterator<int>(0), point_idx(1, dim))
            ), 
        centroids.begin(), centroid_mean<DataType>());
    
    
    /*
    for(int i = 0; i < cluster_size.size(); i++)
        std::cout << "d[" << i << "] = " << cluster_size[i] << std::endl;
    */
    
}


void kmean_thrust(kmean_args_t* args, double* timeMeasure) {
    int n_vals = args->n_vals;
    int n_centroids = args->num_cluster;
    int dims = args->dims;
    args->iter_converge = 0;
    hipEvent_t start, stop, assignStart, assignEnd, updateStart, updateEnd;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&assignStart);
    hipEventCreate(&assignEnd);
    hipEventCreate(&updateStart);
    hipEventCreate(&updateEnd);
    hipEventRecord(start);

    thrust::device_vector<DataType> points(args->input_vals, args->input_vals+n_vals*dims);
    thrust::device_vector<DataType> centroids(args->centroids, args->centroids+n_centroids*dims);
    thrust::device_vector<int> cluster_point(args->clusterId_of_point, args->clusterId_of_point+n_vals);
    thrust::device_vector<int> cluster_size(n_centroids);

    thrust::host_vector<DataType> oldCentroid(n_centroids*dims);
    //thrust::host_vector<DataType> curCentroids(args->centroids, args->centroids+n_centroids*dims);
    
    //thrust::host_vector<int> host_cluster_point(args->clusterId_of_point, args->clusterId_of_point+n_vals);
    thrust::host_vector<DataType> h_centroids(centroids);
    
    while (args->iter_converge < args->max_num_iter && !isConverged(h_centroids, oldCentroid, args->threshold, n_centroids, dims)) {
        args->iter_converge++;
        thrust::copy(h_centroids.begin(), h_centroids.end(), oldCentroid.begin());
        //copyCentroids(tmpCentroid, args->centroids, args->num_cluster, args->dims);
        //memcpy(tmpCentroid, args->centroids, args->n_vals*args->dims);
        hipEventRecord(assignStart);

        assign_centroid(cluster_point, points, centroids, n_vals, dims, n_centroids);

        hipEventRecord(assignEnd);
        hipEventSynchronize(assignEnd);
        float tmp1 = 0;
        hipEventElapsedTime(&tmp1, assignStart, assignEnd);
        timeMeasure[0] += tmp1;

        
        //std::cout<<args->iter_converge<<std::endl;

        thrust::fill(centroids.begin(), centroids.end(), 0.0);
        thrust::fill(cluster_size.begin(), cluster_size.end(), 0);
        
        hipEventRecord(updateStart);
        update_centroid(centroids, cluster_size, cluster_point, points, n_vals, dims, n_centroids);

        hipEventRecord(updateEnd);
        hipEventSynchronize(updateEnd);
        float tmp2 = 0;
        hipEventElapsedTime(&tmp2, updateStart, updateEnd);
        timeMeasure[1] += tmp2;
        //curCentroids = centroids;
        /*
        std::cout<<"Centroids"<<std::endl;
        for(int i = 0; i < n_centroids; i++){
            for (int j = 0; j < dims; j++)
                std::cout << centroids[i*dims+j] <<" ";
            std::cout<<std::endl;
            
        }
        */
        h_centroids = centroids;
    }
    
    
    thrust::host_vector<int> h_cluster_point(cluster_point);
    /*
    for (int clusterId = 0; clusterId < args->num_cluster; clusterId++){
            printf("%d ", clusterId);
            for (int d = 0; d < args->dims; d++)
                printf("%f ", h_centroids[clusterId*args->dims + d]);
        printf("\n");
        }
    */
    //free(args->centroids);
    //free(args->clusterId_of_point);
    for (int i = 0; i < h_centroids.size(); i++){
        args->centroids[i] = h_centroids[i];
    }
    for (int i = 0; i < h_cluster_point.size(); i++){
        args->clusterId_of_point[i] = h_cluster_point[i];
    }
    //args->centroids = thrust::raw_pointer_cast(h_centroids.data());
    //args->clusterId_of_point = thrust::raw_pointer_cast(h_cluster_point.data());

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float tmp3 = 0;
    hipEventElapsedTime(&tmp3, start, stop);
    timeMeasure[2] += tmp3;
    timeMeasure[3] = timeMeasure[2] - timeMeasure[1] - timeMeasure[0];
}
