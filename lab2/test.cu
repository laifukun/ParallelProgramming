#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>

//#define N 64
#define TPB 32
#define K 3
#define MAX_ITER 10

__device__ float distance(float x1, float x2)
{
	return sqrt((x2-x1)*(x2-x1));
}

__global__ void kMeansClusterAssignment(float *d_datapoints, int *d_clust_assn, float *d_centroids, int N)
{
	//get idx for this datapoint
	const int idx = blockIdx.x*blockDim.x + threadIdx.x;

	//bounds check
	if (idx >= N) return;

	//find the closest centroid to this datapoint
	float min_dist = INFINITY;
	int closest_centroid = 0;

	for(int c = 0; c<K;++c)
	{
		float dist = distance(d_datapoints[idx],d_centroids[c]);

		if(dist < min_dist)
		{
			min_dist = dist;
			closest_centroid=c;
		}
	}

	//assign closest cluster id for this datapoint/thread
	d_clust_assn[idx]=closest_centroid;
}


__global__ void kMeansCentroidUpdate(float *d_datapoints, int *d_clust_assn, float *d_centroids, int *d_clust_sizes, int N)
{

	//get idx of thread at grid level
	const int idx = blockIdx.x*blockDim.x + threadIdx.x;

	//bounds check
	if (idx >= N) return;

	//get idx of thread at the block level
	const int s_idx = threadIdx.x;

	//put the datapoints and corresponding cluster assignments in shared memory so that they can be summed by thread 0 later
	__shared__ float s_datapoints[TPB];
	s_datapoints[s_idx]= d_datapoints[idx];

	__shared__ int s_clust_assn[TPB];
	s_clust_assn[s_idx] = d_clust_assn[idx];

	__syncthreads();

	//it is the thread with idx 0 (in each block) that sums up all the values within the shared array for the block it is in
	if(s_idx==0)
	{
		float b_clust_datapoint_sums[K]={0};
		int b_clust_sizes[K]={0};

		for(int j=0; j< blockDim.x; ++j)
		{
			int clust_id = s_clust_assn[j];
			b_clust_datapoint_sums[clust_id]+=s_datapoints[j];
			b_clust_sizes[clust_id]+=1;
		}

		//Now we add the sums to the global centroids and add the counts to the global counts.
		for(int z=0; z < K; ++z)
		{
			atomicAdd(&d_centroids[z],b_clust_datapoint_sums[z]);
			atomicAdd(&d_clust_sizes[z],b_clust_sizes[z]);
		}
	}

	__syncthreads();

	//currently centroids are just sums, so divide by size to get actual centroids
	if(idx < K){
		d_centroids[idx] = d_centroids[idx]/d_clust_sizes[idx]; 
	}

}


int main()
{

	//allocate memory on the device for the data points
	float *d_datapoints=0;
	//allocate memory on the device for the cluster assignments
	int *d_clust_assn = 0;
	//allocate memory on the device for the cluster centroids
	float *d_centroids = 0;
	//allocate memory on the device for the cluster sizes
	int *d_clust_sizes=0;

	int N = 64;
	hipMalloc(&d_datapoints, N*sizeof(float));
	hipMalloc(&d_clust_assn,N*sizeof(int));
	hipMalloc(&d_centroids,K*sizeof(float));
	hipMalloc(&d_clust_sizes,K*sizeof(float));

	float *h_centroids = (float*)malloc(K*sizeof(float));
	float *h_datapoints = (float*)malloc(N*sizeof(float));
	int *h_clust_sizes = (int*)malloc(K*sizeof(int));

	srand(time(0));

	//initialize centroids
	for(int c=0;c<K;++c)
	{
		h_centroids[c]=(float) rand() / (DataType)RAND_MAX;
		printf("%f\n", h_centroids[c]);
		h_clust_sizes[c]=0;
	}

	//initalize datapoints
	for(int d = 0; d < N; ++d)
	{
		h_datapoints[d] = (float) rand() / (DataType)RAND_MAX;
	}

	hipMemcpy(d_centroids,h_centroids,K*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_datapoints,h_datapoints,N*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_clust_sizes,h_clust_sizes,K*sizeof(int),hipMemcpyHostToDevice);

	int cur_iter = 1;

	while(cur_iter < MAX_ITER)
	{
		//call cluster assignment kernel
		kMeansClusterAssignment<<<(N+TPB-1)/TPB,TPB>>>(d_datapoints,d_clust_assn,d_centroids, N);

		//copy new centroids back to host 
		hipMemcpy(h_centroids,d_centroids,K*sizeof(float),hipMemcpyDeviceToHost);

		for(int i =0; i < K; ++i){
			printf("Iteration %d: centroid %d: %f\n",cur_iter,i,h_centroids[i]);
		}

		//reset centroids and cluster sizes (will be updated in the next kernel)
		hipMemset(d_centroids,0.0,K*sizeof(float));
		hipMemset(d_clust_sizes,0,K*sizeof(int));

		//call centroid update kernel
		kMeansCentroidUpdate<<<(N+TPB-1)/TPB,TPB>>>(d_datapoints,d_clust_assn,d_centroids,d_clust_sizes, N);

		cur_iter+=1;
	}

	hipFree(d_datapoints);
	hipFree(d_clust_assn);
	hipFree(d_centroids);
	hipFree(d_clust_sizes);

	free(h_centroids);
	free(h_datapoints);
	free(h_clust_sizes);

	return 0;
}