#include "hip/hip_runtime.h"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include "kmeans.h"
#include "defines.h"


template <typename T>
struct index_to_row : public thrust::unary_function<T,T>
{
  T C; // number of columns
  
  __host__ __device__
  index_to_row(T C) : C(C) {}

  __host__ __device__
  T operator()(T i)
  {
    return i / C;
  }
};

template <typename T>
struct square
{
    __host__ __device__
        T operator()(const T& x) const { 
            return x * x;
        }
};

template <typename T>
struct lessThanThreshold
{
    T threshold; 
  
    __host__ __device__
    lessThanThreshold(T t) : threshold(t) {}

    __host__ __device__
        bool operator()(const T& y) const { 
            //printf("y: %e, sqrt(y): %e\n", y, sqrt(y));
            //return y >= threshold*threshold ? false : true;
            return (y >= threshold) ? false : true;
        }
};

template <typename T>
struct squareDiff
{
    __host__ __device__
        T operator()(const T& x, const T& y) const { 
            return (x-y) * (x-y);
        }
};

struct boolAnd
{
    __host__ __device__
        bool operator()(const bool x, const bool y) const { 
            return x&&y;
        }
};

typedef thrust::tuple<DataType, DataType> point_center_tuple;

struct point_distance : public thrust::unary_function<point_center_tuple, DataType>
{
  __host__ __device__ DataType operator()(const point_center_tuple &pc_tuple) const {
    DataType temp = thrust::get<0>(pc_tuple) - thrust::get<1>(pc_tuple);
    return temp*temp;
  }
};


struct point_idx
{
  int dim;
  int n_c;

  point_idx(int _dim, int _n) : dim(_dim), n_c(_n) {};

  __host__ __device__ int operator()(const int idx) const {
      return (idx % dim) + (dim * (idx/(dim*n_c)));    
    }
};

struct center_idx
{
  int dim;
  int n_c;

  center_idx(int _dim, int _n) : dim(_dim), n_c(_n) {};

  __host__ __device__ int operator()(const int idx) const {
    return (idx % (dim*n_c));
    }
};

template <typename T>
struct my_sqrt
{
  __host__ __device__ T operator()(const T val) const {
    return sqrtf(val);
  }
};

template<typename T>
struct min_index
{
    __host__ __device__
        T operator()(const T& x, const T& y) const { 
            return (x-y) * (x-y);
        }
};


struct size_accumulator {
  int *r;
  size_accumulator(int *_r) : r(_r) {};
  
  template <typename T>
  __host__ __device__
  void operator()(T t) {
    atomicAdd(r+thrust::get<0>(t), thrust::get<1>(t));  
  }
};

struct centroid_accumulator {
  DataType *c, *p;
  int dim;
  centroid_accumulator(DataType *_c, DataType *_p, int _dim) : c(_c), p(_p), dim(_dim) {};
  
  template <typename T>
  __host__ __device__
  void operator()(T t) {
      //printf("val: %d, idx: %d \n", thrust::get<0>(t), thrust::get<1>(t));
      for (int i = 0; i < dim; i++)
        atomicAdd(c+thrust::get<0>(t)*dim + i, *(p + thrust::get<1>(t)*dim + i));  
  }
};

template <typename T>
struct centroid_mean {

  __host__ __device__
  T operator()(const T& val, const int& count) {
      //printf("val: %lf count: %d\n", val, count);
      //if (count == 0) return val;
      return val/count;
  }
};

__host__ bool isConverged(thrust::host_vector<DataType>& curCentroid, thrust::host_vector<DataType>& prevCentroid, DataType threshold, int n_centroids, int dim) {
    thrust::host_vector<DataType> point_diff(n_centroids);
    //thrust::host_vector<int> point_idx(n);
    thrust::host_vector<DataType> tmp(n_centroids*dim);
    thrust::transform(curCentroid.begin(), curCentroid.end(), prevCentroid.begin(), tmp.begin(), squareDiff<DataType>());
    
    thrust::reduce_by_key
    (thrust::make_transform_iterator(thrust::counting_iterator<int>(0), index_to_row<int>(dim)),
     thrust::make_transform_iterator(thrust::counting_iterator<int>(n_centroids*dim), index_to_row<int>(dim)),
     tmp.begin(),
     thrust::make_discard_iterator(),
     //point_idx.begin(),
     point_diff.begin(),
     thrust::equal_to<int>(),
     thrust::plus<DataType>());

    thrust::transform(point_diff.begin(), point_diff.end(), point_diff.begin(), my_sqrt<DataType>());
    lessThanThreshold<DataType> ltThreshold(threshold);
    
    
    /*
    for(int i = 0; i < point_diff.size(); i++)
        std::cout << "p[" << i << "] = " << point_diff[i] << std::endl;
    */

    return thrust::transform_reduce(point_diff.begin(), point_diff.end(), ltThreshold, true,  boolAnd());
}


void assign_centroid(thrust::device_vector<int>& cluster_point,
                    thrust::device_vector<DataType>& points, 
                    thrust::device_vector<DataType>& centroids, 
                    int n_vals, int dim, int n_centroids) {

        thrust::device_vector<DataType> dist(n_vals*n_centroids);
          thrust::reduce_by_key(
              thrust::make_transform_iterator(thrust::make_counting_iterator<int>(0), index_to_row<int>(dim)), 
              thrust::make_transform_iterator(thrust::make_counting_iterator<int>(dim*n_vals*n_centroids), index_to_row<int>(dim)
                  ),
              thrust::make_transform_iterator(
                thrust::make_zip_iterator(
                    thrust::make_tuple(
                                  thrust::make_permutation_iterator(
                                              points.begin(), 
                                              thrust::make_transform_iterator(
                                                  thrust::make_counting_iterator<int>(0), 
                                                  point_idx(dim, n_centroids)
                                                  )
                                            ),
                                  thrust::make_permutation_iterator(
                                      centroids.begin(), 
                                      thrust::make_transform_iterator(
                                          thrust::make_counting_iterator<int>(0), 
                                          center_idx(dim, n_centroids)
                                          )
                                    )                                    
                                )
                            ), 
                            point_distance()
                        ), 
              thrust::make_discard_iterator(), 
              dist.begin()
            );
    /*
    std::cout<<"size "<< dist.size()<<std::endl;
    std::cout<<std::endl;
    for(int i = 0; i < n_vals; i++){
        for (int j = 0; j < n_centroids; j++)
            std::cout << dist[i*n_centroids+j] <<" ";
        std::cout<<std::endl;
    }
    
    std::cout<<"Centroids"<<std::endl;
    for(int i = 0; i < n_centroids; i++){
        for (int j = 0; j < dim; j++)
            std::cout << centroids[i*dim+j] <<" ";
        std::cout<<std::endl;
    }
    */

/*
    std::cout<<"Points"<<std::endl;
    for(int i = 0; i < n_centroids; i++){
        for (int j = 0; j < dim; j++)
            std::cout << points[i*dim+j] <<" ";
        std::cout<<std::endl;
    }
    */
    thrust::device_vector<DataType> minval(n_vals);
    thrust::reduce_by_key
    (thrust::make_transform_iterator(thrust::counting_iterator<int>(0), index_to_row<int>(n_centroids)),
     thrust::make_transform_iterator(thrust::counting_iterator<int>(n_vals*n_centroids), index_to_row<int>(n_centroids)),
     thrust::make_zip_iterator(
            thrust::make_tuple(
                /*thrust::make_permutation_iterator(
                dist.begin(),
                thrust::make_transform_iterator(
                    thrust::make_counting_iterator<int>(0), 
                    //thrust::placeholders::_1 % n_centroids
                    //(thrust::placeholders::_1 % n_vals) * n_centroids + thrust::placeholders::_1 / n_vals
                    //(thrust::placeholders::_1 % n_centroids) * n_vals + thrust::placeholders::_1 / n_centroids
                    //thrust::placeholders::_1 / n_centroids
                    //point_idx(1, n_centroids)
                    )
                ),*/
                dist.begin(),
                thrust::make_transform_iterator(thrust::make_counting_iterator<int>(0), 
                thrust::placeholders::_1 % n_centroids
                //center_idx(1, n_centroids)                
                )
                )
        ),
     thrust::make_discard_iterator(),
     thrust::make_zip_iterator(
                    thrust::make_tuple(
                            minval.begin(),
                            cluster_point.begin())),
     thrust::equal_to<int>(),
     thrust::minimum<thrust::tuple<DataType, int> >()
     );
    /*
     for(int i = 0; i < cluster_point.size(); i++)
        std::cout << "c[" << i << "] = " << minval[i] << " "<<  cluster_point[i] << std::endl;
        */

}


void update_centroid(thrust::device_vector<DataType>& centroids, 
                    thrust::device_vector<int>& cluster_size,
                    thrust::device_vector<int>& cluster_point,
                    thrust::device_vector<DataType>& points,
                    int n_vals, int dim, int n_centroids) {

    //for(int i = 0; i < cluster_point.size(); i++)
    //    std::cout << "d[" << i << "] = " << cluster_point[i] << std::endl;
    /*
    thrust::transform(thrust::make_permutation_iterator(cluster_size.begin(), cluster_point.begin()), 
        thrust::make_permutation_iterator(cluster_size.begin(), cluster_point.end()), 
        thrust::make_permutation_iterator(cluster_size.begin(), cluster_point.begin()),
        add_one());
    */
   //thrust::device_vector<DataType> tmp_points(points);
   //thrust::sort_by_key(cluster_point.begin(), cluster_point.end(), tmp_points.begin());
   //thrust::reduce_by_key(cluster_point.begin(), cluster_point.end(), thrust::constant_iterator<int>(1), thrust::make_discard_iterator(),cluster_size.begin());
    //thrust::for_each_n(thrust::make_zip_iterator(thrust::make_tuple(d_a.begin(), d_b.begin())), size_a, my_func(thrust::raw_pointer_cast(d_r.data())));
    thrust::for_each_n(thrust::make_zip_iterator(thrust::make_tuple(cluster_point.begin(), thrust::constant_iterator<int>(1))), 
                n_vals, size_accumulator(thrust::raw_pointer_cast(cluster_size.data())));
    
    thrust::for_each_n(thrust::make_zip_iterator(thrust::make_tuple(cluster_point.begin(), thrust::counting_iterator<int>(0))), 
                n_vals, centroid_accumulator(thrust::raw_pointer_cast(centroids.data()), thrust::raw_pointer_cast(points.data()), dim));
    
    //thrust::transform_iterator<linear_index_to_row_index<int>, thrust::counting_iterator<int>> centroids_first(centroids, linear_index_to_row_index<int>(dim));
    thrust::transform(centroids.begin(), centroids.end(), 
        thrust::make_permutation_iterator(
                cluster_size.begin(), 
                thrust::make_transform_iterator(thrust::make_counting_iterator<int>(0), point_idx(1, dim))
            ), 
        centroids.begin(), centroid_mean<DataType>());
    
    
    /*
    for(int i = 0; i < cluster_size.size(); i++)
        std::cout << "d[" << i << "] = " << cluster_size[i] << std::endl;
    */
    
}


void kmean_thrust(kmean_args_t* args, double* timeMeasure) {
    int n_vals = args->n_vals;
    int n_centroids = args->num_cluster;
    int dims = args->dims;
    args->iter_converge = 0;
    hipEvent_t start, stop, assignStart, assignEnd, updateStart, updateEnd;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&assignStart);
    hipEventCreate(&assignEnd);
    hipEventCreate(&updateStart);
    hipEventCreate(&updateEnd);
    hipEventRecord(start);

    thrust::device_vector<DataType> points(args->input_vals, args->input_vals+n_vals*dims);
    thrust::device_vector<DataType> centroids(args->centroids, args->centroids+n_centroids*dims);
    thrust::device_vector<int> cluster_point(args->clusterId_of_point, args->clusterId_of_point+n_vals);
    thrust::device_vector<int> cluster_size(n_centroids);

    thrust::host_vector<DataType> oldCentroid(n_centroids*dims);
    //thrust::host_vector<DataType> curCentroids(args->centroids, args->centroids+n_centroids*dims);
    
    //thrust::host_vector<int> host_cluster_point(args->clusterId_of_point, args->clusterId_of_point+n_vals);
    thrust::host_vector<DataType> h_centroids(centroids);
    
    while (args->iter_converge < args->max_num_iter && !isConverged(h_centroids, oldCentroid, args->threshold, n_centroids, dims)) {
        args->iter_converge++;
        thrust::copy(h_centroids.begin(), h_centroids.end(), oldCentroid.begin());
        //copyCentroids(tmpCentroid, args->centroids, args->num_cluster, args->dims);
        //memcpy(tmpCentroid, args->centroids, args->n_vals*args->dims);
        hipEventRecord(assignStart);

        assign_centroid(cluster_point, points, centroids, n_vals, dims, n_centroids);

        hipEventRecord(assignEnd);
        hipEventSynchronize(assignEnd);
        float tmp1 = 0;
        hipEventElapsedTime(&tmp1, assignStart, assignEnd);
        timeMeasure[0] += tmp1;

        
        //std::cout<<args->iter_converge<<std::endl;

        thrust::fill(centroids.begin(), centroids.end(), 0.0);
        thrust::fill(cluster_size.begin(), cluster_size.end(), 0);
        
        hipEventRecord(updateStart);
        update_centroid(centroids, cluster_size, cluster_point, points, n_vals, dims, n_centroids);

        hipEventRecord(updateEnd);
        hipEventSynchronize(updateEnd);
        float tmp2 = 0;
        hipEventElapsedTime(&tmp2, updateStart, updateEnd);
        timeMeasure[1] += tmp2;
        //curCentroids = centroids;
        /*
        std::cout<<"Centroids"<<std::endl;
        for(int i = 0; i < n_centroids; i++){
            for (int j = 0; j < dims; j++)
                std::cout << centroids[i*dims+j] <<" ";
            std::cout<<std::endl;
            
        }
        */
        h_centroids = centroids;
    }
    
    
    thrust::host_vector<int> h_cluster_point(cluster_point);
    /*
    for (int clusterId = 0; clusterId < args->num_cluster; clusterId++){
            printf("%d ", clusterId);
            for (int d = 0; d < args->dims; d++)
                printf("%f ", h_centroids[clusterId*args->dims + d]);
        printf("\n");
        }
    */
    //free(args->centroids);
    //free(args->clusterId_of_point);
    for (int i = 0; i < h_centroids.size(); i++){
        args->centroids[i] = h_centroids[i];
    }
    for (int i = 0; i < h_cluster_point.size(); i++){
        args->clusterId_of_point[i] = h_cluster_point[i];
    }
    //args->centroids = thrust::raw_pointer_cast(h_centroids.data());
    //args->clusterId_of_point = thrust::raw_pointer_cast(h_cluster_point.data());

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float tmp3 = 0;
    hipEventElapsedTime(&tmp3, start, stop);
    timeMeasure[2] += tmp3;
    timeMeasure[3] = timeMeasure[2] - timeMeasure[1] - timeMeasure[0];
}
